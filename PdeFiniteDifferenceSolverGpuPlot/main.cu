#include "hip/hip_runtime.h"
// Main.cpp : Defines the entry point for the console application.
//

#include <fstream>
#include <algorithm>
#include <chrono>

#include <forge.h>
#define USE_FORGE_CUDA_COPY_HELPERS
#include <ComputeCopy.h>
#include <complex>
#include <cmath>
#include <vector>
#include <iostream>

#include <Vector.h>
#include <ColumnWiseMatrix.h>

#include <AdvectionDiffusionSolver1D.h>
#include <AdvectionDiffusionSolver2D.h>
#include <WaveEquationSolver1D.h>
#include <WaveEquationSolver2D.h>
#include <IterableEnum.h>

#pragma region Command Line Parser

class CommandLineArgumentParser
{
public:
	CommandLineArgumentParser(int argc, char **argv)
		: args(argv, argv + argc)
	{
	}

	template<typename T>
	T GetArgumentValue(const std::string& option) const;

	template<typename T>
	T GetArgumentValue(const std::string& option, const T& defaultValue) const noexcept
	{
		T ret;
		try
		{
			ret = GetArgumentValue<T>(option);
		}
		catch (int)
		{
			ret = defaultValue;
		}

		return ret;
	}

	bool GetFlag(const std::string& option) const
	{
		return std::find(args.begin(), args.end(), option) != args.end();
	}

private:
	std::vector<std::string> args;
};

template<>
std::string CommandLineArgumentParser::GetArgumentValue<std::string>(const std::string& option) const
{
	auto itr = std::find(args.begin(), args.end(), option);
	if (itr != args.end())
	{
		if (++itr == args.end())
			std::abort();
		return *itr;
	}

	throw 42;
}

template<>
int CommandLineArgumentParser::GetArgumentValue<int>(const std::string& option) const
{
	return std::atoi(GetArgumentValue<std::string>(option).c_str());
}

template<>
double CommandLineArgumentParser::GetArgumentValue<double>(const std::string& option) const
{
	return std::atof(GetArgumentValue<std::string>(option).c_str());
}

#pragma endregion

#pragma region Enum Mapping

#define PARSE(E, X)\
	if (!strcmp(text.c_str(), #X))\
		return E::X;

SolverType parseSolverType(const std::string& text)
{
#define PARSE_WORKER(X) PARSE(SolverType, X);

	PARSE_WORKER(ExplicitEuler);
	PARSE_WORKER(ImplicitEuler);
	PARSE_WORKER(CrankNicolson);
	PARSE_WORKER(RungeKuttaRalston);
	PARSE_WORKER(RungeKutta3);
	PARSE_WORKER(RungeKutta4);
	PARSE_WORKER(RungeKuttaThreeEight);
	PARSE_WORKER(RungeKuttaGaussLegendre4);
	PARSE_WORKER(RichardsonExtrapolation2);
	PARSE_WORKER(RichardsonExtrapolation3);
	PARSE_WORKER(AdamsBashforth2);
	PARSE_WORKER(AdamsMouldon2);

#undef PARSE_WORKER

	return SolverType::Null;
}

SpaceDiscretizerType parseSpaceDiscretizer(const std::string& text)
{
#define PARSE_WORKER(X) PARSE(SpaceDiscretizerType, X);

	PARSE_WORKER(Centered);
	PARSE_WORKER(Upwind);
	PARSE_WORKER(LaxWendroff);

#undef PARSE_WORKER
	return SpaceDiscretizerType::Null;
}

BoundaryConditionType parseBoundaryConditionType(const std::string& text)
{
#define PARSE_WORKER(X) PARSE(BoundaryConditionType, X);

	PARSE_WORKER(Dirichlet);
	PARSE_WORKER(Neumann);
	PARSE_WORKER(Periodic);

#undef PARSE_WORKER
	return BoundaryConditionType::Null;
}

MathDomain parseMathDomain(const std::string& text)
{
#define PARSE_WORKER(X) PARSE(MathDomain, X);

	PARSE_WORKER(Double);
	PARSE_WORKER(Float);

#undef PARSE_WORKER
	return MathDomain::Null;
}

#undef PARSE

#pragma endregion

template<class solverImpl, MathDomain md>
void runner1D(const CommandLineArgumentParser& ap, const bool debug)
{
	std::chrono::time_point<std::chrono::high_resolution_clock> start, end;

#define DEBUG_PRINT_START(X)\
	if (debug)\
    {\
	    start = std::chrono::high_resolution_clock::now(); \
		std::cout << #X << std::endl;\
	}

#define DEBUG_PRINT_END\
	if (debug)\
	{\
		end = std::chrono::high_resolution_clock::now(); \
		double elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();\
		std::cout << "... Done" << "[ " << elapsedTime << " ms ]" << std::endl;\
	}

#define DEBUG_PRINT(X)\
	if (debug)\
        std::cout << #X << std::endl;

	DEBUG_PRINT_START(Parsing inputs ...)

#pragma region Parse Inputs

	auto initialConditionFileString = ap.GetArgumentValue<std::string>("-ic", "");
	auto gridFileString = ap.GetArgumentValue<std::string>("-g", "");
	auto outputFileString = ap.GetArgumentValue<std::string>("-of", "sol.cl");

	auto leftBoundaryConditionTypeString = ap.GetArgumentValue<std::string>("-lbct");
	auto leftBoundaryConditionType = parseBoundaryConditionType(leftBoundaryConditionTypeString);
	double leftBoundaryConditionValue = ap.GetArgumentValue<double>("-lbc");

	std::string rightBoundaryConditionTypeString = ap.GetArgumentValue("-rbct", leftBoundaryConditionTypeString);
	auto rightBoundaryConditionType = parseBoundaryConditionType(rightBoundaryConditionTypeString);
	double rightBoundaryConditionValue = ap.GetArgumentValue("-rbc", leftBoundaryConditionValue);

	auto solverType = parseSolverType(ap.GetArgumentValue<std::string>("-st", "CrankNicolson"));
	auto spaceDiscretizerType = parseSpaceDiscretizer(ap.GetArgumentValue<std::string>("-sdt", "Upwind"));
	if (spaceDiscretizerType == SpaceDiscretizerType::LaxWendroff && solverType != SolverType::ExplicitEuler)
	{
		std::cout << "WARNING: Lax-Wendroff scheme can be applied only with ExplicitEuler -> overriding solver type" << std::endl;
		solverType = SolverType::ExplicitEuler;
	}

	auto diffusion = ap.GetArgumentValue<double>("-d");
	auto velocity = ap.GetArgumentValue<double>("-v");
	auto dt = ap.GetArgumentValue<double>("-dt");

	// steps to advance before outputing the solution
	auto n = ap.GetArgumentValue<int>("-n");

	// total number of steps
	size_t N = static_cast<size_t>(ap.GetArgumentValue<int>("-N"));

#pragma endregion

	DEBUG_PRINT_END;

	using vType = cl::Vector<MemorySpace::Device, md>;
	using sType = typename vType::stdType;

	DEBUG_PRINT_START(Creating grid...);

	std::ifstream gridFile(gridFileString);
	vType *grid = nullptr;
	if (!gridFile.is_open())
	{
		DEBUG_PRINT(... creating linspace(-4, 4, 128));
		grid = new vType(cl::LinSpace<MemorySpace::Device, md>(sType(-4.0), sType(4.0), 128u));
	}
	else
	{
		DEBUG_PRINT(... reading from file);
		grid = new vType(cl::VectorFromBinaryFile<MemorySpace::Device, md>(gridFileString));
	}
	DEBUG_PRINT_END;

	DEBUG_PRINT_START(Creating initial condition ...);

	std::ifstream initialConditionFile(initialConditionFileString);
	vType *initialCondition = nullptr;
	if (!initialConditionFile.is_open())
	{
		DEBUG_PRINT(... creating bell function);
		auto _grid = grid->Get();
		std::vector<sType> bellFunction(grid->size());
		for (unsigned i = 0; i < bellFunction.size(); ++i)
			bellFunction[i] = exp(-.25 * _grid[i] * _grid[i]);

		initialCondition = new vType(bellFunction);
	}
	else
	{
		DEBUG_PRINT(... reading from file);
		initialCondition = new vType(cl::VectorFromBinaryFile<MemorySpace::Device, md>(initialConditionFileString));
	}
	DEBUG_PRINT_END;

	BoundaryCondition leftBc(leftBoundaryConditionType, leftBoundaryConditionValue);
	BoundaryCondition rightBc(rightBoundaryConditionType, rightBoundaryConditionValue);
	BoundaryCondition1D bc(leftBc, rightBc);

	DEBUG_PRINT_START(Creating PDE input data ...);
	pde::PdeInputData1D<MemorySpace::Device, md> data(*initialCondition, *grid, velocity, diffusion, dt, solverType, spaceDiscretizerType, bc);
	DEBUG_PRINT_END;

	DEBUG_PRINT_START(Creating PDE solver ...);
	solverImpl solver(data);
	DEBUG_PRINT_END;

	std::vector<sType> solutionMatrix;

	DEBUG_PRINT_START(Solving ...);

	forge::Window wnd(1000, 800, "Plotting Demo");
	wnd.makeCurrent();

	forge::Chart chart(FG_CHART_2D);
	auto _grid = grid->Get();
	auto _initialCondition = initialCondition->Get();
	chart.setAxesLimits(_grid.front(), _grid.back(), *std::min_element(_initialCondition.begin(), _initialCondition.end()), *std::max_element(_initialCondition.begin(), _initialCondition.end()));

	static constexpr forge::dtype precision = forge::f32;
	forge::Plot plt = chart.plot(grid->size(), precision, FG_PLOT_LINE, FG_MARKER_NONE);
	plt.setColor(FG_BLUE);

	GfxHandle* handles;
	createGLBuffer(&handles, plt.vertices(), FORGE_VERTEX_BUFFER);

	bool toDo = true;
	cl::Vector<MemorySpace::Device, MathDomain::Float>* xyPair = nullptr;
	do
	{		
		if (toDo)
		{
			for (unsigned m = 0; m < N; ++m)
			{
				solver.Advance(n);

				if (!xyPair)
					xyPair = new cl::Vector<MemorySpace::Device, MathDomain::Float>(2 * grid->size());
				cl::MakePair(*xyPair, *grid, *solver.solution->columns[0]);
				copyToGLBuffer(handles, (ComputeResourceHandle)xyPair->GetBuffer().pointer, plt.verticesSize());
				wnd.draw(chart);
			}
		}

		wnd.draw(chart);
		toDo = false;
	}
	while (!wnd.close());
	releaseGLBuffer(handles);

	DEBUG_PRINT_END;

	delete initialCondition;
	delete grid;

#undef DEBUB_PRINT
}

template<class solverImpl, MathDomain md>
void runner2D(const CommandLineArgumentParser& ap, const bool debug)
{
	std::chrono::time_point<std::chrono::high_resolution_clock> start, end;

	DEBUG_PRINT_START(Parsing inputs ...)

#pragma region Parse Inputs

	auto initialConditionFileString = ap.GetArgumentValue<std::string>("-ic", "");
	auto xGridFileString = ap.GetArgumentValue<std::string>("-gx", "");
	auto yGridFileString = ap.GetArgumentValue<std::string>("-gy", "");
	auto outputFileString = ap.GetArgumentValue<std::string>("-of", "sol.cl");

#pragma region BC

	auto leftBoundaryConditionTypeString = ap.GetArgumentValue<std::string>("-lbct");
	auto leftBoundaryConditionType = parseBoundaryConditionType(leftBoundaryConditionTypeString);
	double leftBoundaryConditionValue = ap.GetArgumentValue<double>("-lbc");

	std::string rightBoundaryConditionTypeString = ap.GetArgumentValue("-rbct", leftBoundaryConditionTypeString);
	auto rightBoundaryConditionType = parseBoundaryConditionType(rightBoundaryConditionTypeString);
	double rightBoundaryConditionValue = ap.GetArgumentValue("-rbc", leftBoundaryConditionValue);

	auto downBoundaryConditionTypeString = ap.GetArgumentValue<std::string>("-dbct", leftBoundaryConditionTypeString);
	auto downBoundaryConditionType = parseBoundaryConditionType(downBoundaryConditionTypeString);
	auto downBoundaryConditionValue = ap.GetArgumentValue<double>("-dbc", leftBoundaryConditionValue);

	auto upBoundaryConditionTypeString = ap.GetArgumentValue<std::string>("-ubct", leftBoundaryConditionTypeString);
	auto upBoundaryConditionType = parseBoundaryConditionType(upBoundaryConditionTypeString);
    auto upBoundaryConditionValue = ap.GetArgumentValue<double>("-ubc", leftBoundaryConditionValue);

#pragma endregion

	auto solverType = parseSolverType(ap.GetArgumentValue<std::string>("-st", "CrankNicolson"));
	auto spaceDiscretizerType = parseSpaceDiscretizer(ap.GetArgumentValue<std::string>("-sdt", "Upwind"));
	if (spaceDiscretizerType == SpaceDiscretizerType::LaxWendroff && solverType != SolverType::ExplicitEuler)
	{
		std::cout << "WARNING: Lax-Wendroff scheme can be applied only with ExplicitEuler -> overriding solver type" << std::endl;
		solverType = SolverType::ExplicitEuler;
	}

	auto diffusion = ap.GetArgumentValue<double>("-d");
	auto xVelocity = ap.GetArgumentValue<double>("-vx");
	auto yVelocity = ap.GetArgumentValue<double>("-vy");
	auto dt = ap.GetArgumentValue<double>("-dt");

	// steps to advance before outputing the solution
	auto n = ap.GetArgumentValue<int>("-n");

	// total number of steps
	unsigned N = static_cast<unsigned>(ap.GetArgumentValue<int>("-N"));

#pragma endregion

	DEBUG_PRINT_END;

	using vType = cl::Vector<MemorySpace::Device, md>;
	using mType = cl::ColumnWiseMatrix<MemorySpace::Device, md>;
	using sType = typename vType::stdType;

#pragma region Grid

	DEBUG_PRINT_START(Creating x grid...);

	std::ifstream xGridFile(xGridFileString);
	vType *xGrid = nullptr;
	constexpr unsigned defaultSize = 128u;
	if (!xGridFile.is_open())
	{
		DEBUG_PRINT(... creating linspace(-4, 4, 128));
		xGrid = new vType(cl::LinSpace<MemorySpace::Device, md>(sType(-4.0), sType(4.0), defaultSize));
	}
	else
	{
		DEBUG_PRINT(... reading from file);
		xGrid = new vType(cl::VectorFromBinaryFile<MemorySpace::Device, md>(xGridFileString));
	}
	DEBUG_PRINT_END;

	DEBUG_PRINT_START(Creating y grid...);

	std::ifstream yGridFile(yGridFileString);
	vType *yGrid = nullptr;
	if (!yGridFile.is_open())
	{
		DEBUG_PRINT(... creating linspace(-4, 4, 128));
		yGrid = new vType(cl::LinSpace<MemorySpace::Device, md>(sType(-4.0), sType(4.0), defaultSize));
	}
	else
	{
		DEBUG_PRINT(... reading from file);
		yGrid = new vType(cl::VectorFromBinaryFile<MemorySpace::Device, md>(yGridFileString));
	}
	DEBUG_PRINT_END;

#pragma endregion

	DEBUG_PRINT_START(Creating initial condition ...);

	std::ifstream initialConditionFile(initialConditionFileString);
	mType *initialCondition = nullptr;
	if (!initialConditionFile.is_open())
	{
		DEBUG_PRINT(... creating bell function);
		auto _xGrid = xGrid->Get();
		auto _yGrid = yGrid->Get();
		std::vector<sType> bellFunction(xGrid->size() * yGrid->size());
		for (unsigned j = 0; j < _yGrid.size(); ++j)
			for (unsigned i = 0; i < _xGrid.size(); ++i)
				bellFunction[i + _xGrid.size() * j] = exp(-.25 * (_xGrid[i] * _xGrid[i] + _yGrid[j] * _yGrid[j]));

		initialCondition = new mType(bellFunction, xGrid->size(), yGrid->size());
	}
	else
	{
		DEBUG_PRINT(... reading from file);
		initialCondition = new mType(cl::MatrixFromBinaryFile<MemorySpace::Device, md>(initialConditionFileString));
	}
	DEBUG_PRINT_END;

	BoundaryCondition leftBc(leftBoundaryConditionType, leftBoundaryConditionValue);
	BoundaryCondition rightBc(rightBoundaryConditionType, rightBoundaryConditionValue);
	BoundaryCondition downBc(downBoundaryConditionType, downBoundaryConditionValue);
	BoundaryCondition upBc(upBoundaryConditionType, upBoundaryConditionValue);
	BoundaryCondition2D bc(leftBc, rightBc, downBc, upBc);

	DEBUG_PRINT_START(Creating PDE input data ...);
	pde::PdeInputData2D<MemorySpace::Device, md> data(*initialCondition, *xGrid, *yGrid,
													  xVelocity, yVelocity, diffusion, dt, solverType, spaceDiscretizerType, bc);
	DEBUG_PRINT_END;

	DEBUG_PRINT_START(Creating PDE solver ...);
	solverImpl solver(data);
	DEBUG_PRINT_END;

	// solution matrix is a collection of flattened solutions over time
	forge::Window wnd(1024, 768, "3d Surface Demo");
	wnd.makeCurrent();

	forge::Chart chart(FG_CHART_3D);

	auto _xGrid = xGrid->Get();
	auto _yGrid = yGrid->Get();
	auto _ic = initialCondition->Get();
	chart.setAxesLimits(_xGrid.front(), _xGrid.back(), _yGrid.front(), _yGrid.back(), *std::min_element(_ic.begin(), _ic.end()), *std::max_element(_ic.begin(), _ic.end()));
	chart.setAxesTitles("x-axis", "y-axis", "z-axis");

	forge::Surface surf = chart.surface(_xGrid.size(), _yGrid.size(), forge::f32);
	surf.setColor(FG_BLUE);

	GfxHandle* handle;
	createGLBuffer(&handle, surf.vertices(), FORGE_VERTEX_BUFFER);

	bool toDo = true;
	cl::Vector<MemorySpace::Device, MathDomain::Float>* xyzTriple = nullptr;
	do
	{
		if (toDo)
		{
			for (unsigned m = 0; m < N; ++m)
			{
				solver.Advance(n);

				if (!xyzTriple)
					xyzTriple = new cl::Vector<MemorySpace::Device, MathDomain::Float>(3 * xGrid->size() * yGrid->size());

				cl::MakeTriple(*xyzTriple, *xGrid, *yGrid, *solver.solution->columns[0]);
				copyToGLBuffer(handle, (ComputeResourceHandle)xyzTriple->GetBuffer().pointer, surf.verticesSize());
				wnd.draw(chart);
			}
		}

		wnd.draw(chart);
		toDo = false;
	}
	while (!wnd.close());
	releaseGLBuffer(handle);

	delete initialCondition;
	delete xGrid;
	delete yGrid;

#undef DEBUB_PRINT
}


int main(int argc, char** argv)
{
	CommandLineArgumentParser ap(argc, argv);

	auto mathDomain = parseMathDomain(ap.GetArgumentValue<std::string>("-md", "Float"));
	auto pdeType = ap.GetArgumentValue<std::string>("-pde", "AdvectionDiffusion");
	auto dimensionality = ap.GetArgumentValue<int>("-dim", 1);
	auto debug = ap.GetFlag("-dbg");

	if (dimensionality == 1)
	{
		switch (mathDomain)
		{
			case MathDomain::Float:
				if (pdeType == "AdvectionDiffusion")
					runner1D<pde::AdvectionDiffusionSolver1D<MemorySpace::Device, MathDomain::Float>, MathDomain::Float>(ap, debug);
				else if (pdeType == "WaveEquation")
					runner1D<pde::WaveEquationSolver1D<MemorySpace::Device, MathDomain::Float>, MathDomain::Float>(ap, debug);
				else
					throw NotImplementedException();
				break;
			case MathDomain::Double:
				if (pdeType == "AdvectionDiffusion")
					runner1D<pde::AdvectionDiffusionSolver1D<MemorySpace::Device, MathDomain::Double>, MathDomain::Double>(ap, debug);
				else if (pdeType == "WaveEquation")
					runner1D<pde::WaveEquationSolver1D<MemorySpace::Device, MathDomain::Double>, MathDomain::Double>(ap, debug);
				else
					throw NotImplementedException();
				break;
			default:
				throw NotImplementedException();
		}
	}
	else if (dimensionality == 2)
	{
		switch (mathDomain)
		{
			case MathDomain::Float:
				if (pdeType == "AdvectionDiffusion")
					runner2D<pde::AdvectionDiffusionSolver2D<MemorySpace::Device, MathDomain::Float>, MathDomain::Float>(ap, debug);
				else if (pdeType == "WaveEquation")
					runner2D<pde::WaveEquationSolver2D<MemorySpace::Device, MathDomain::Float>, MathDomain::Float>(ap, debug);
				else
					throw NotImplementedException();
				break;
			case MathDomain::Double:
				if (pdeType == "AdvectionDiffusion")
					runner2D<pde::AdvectionDiffusionSolver2D<MemorySpace::Device, MathDomain::Double>, MathDomain::Double>(ap, debug);
				else if (pdeType == "WaveEquation")
					runner2D<pde::WaveEquationSolver2D<MemorySpace::Device, MathDomain::Double>, MathDomain::Double>(ap, debug);  // FIXME
				else
					throw NotImplementedException();
				break;
			default:
				throw NotImplementedException();
		}
	}

	return 0;
}

